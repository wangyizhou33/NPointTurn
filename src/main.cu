#include "Paper.hpp"
#include <iostream>

int main(void)
{
    uint32_t* dev_reach[ITER_CNT];
    uint32_t* reach[ITER_CNT];
    uint32_t* dev_fb;

    // setup
    for (uint32_t iter = 0; iter < ITER_CNT; ++iter)
    {
        HANDLE_ERROR(hipMalloc((void**)&dev_reach[iter], SIZE));
        HANDLE_ERROR(hipMemset((void*)dev_reach[iter], 0, SIZE));

        reach[iter] = (uint32_t*)malloc(SIZE);
        memset((void*)reach[iter], 0, SIZE);
    }

    HANDLE_ERROR(hipMalloc((void**)&dev_fb, SIZE));
    HANDLE_ERROR(hipMemset((void*)dev_fb, 2147483647, SIZE)); // set all ones

    // set start
    uint32_t origin = turnCoord(X_DIM / 2, Y_DIM / 2, 0,
                                X_DIM, Y_DIM, POS_RES, HDG_RES, TURN_R);
    bitVectorWrite(reach[0], 1, origin);

    // host to device
    HANDLE_ERROR(hipMemcpy(dev_reach[0], reach[0], SIZE,
                            hipMemcpyHostToDevice));

    TIME_PRINT("search: ",
               for (uint32_t iter = 0; iter + 2 < ITER_CNT; iter += 2) {
                   bitSweepLeft(dev_reach[iter + 1],
                                dev_fb,
                                dev_reach[iter],
                                TURN_R,
                                nullptr);

                   bitSweepLeft(dev_reach[iter + 2],
                                dev_fb,
                                dev_reach[iter + 1],
                                -TURN_R,
                                nullptr);
               } HANDLE_ERROR(hipDeviceSynchronize()););
    for (uint32_t iter = 0; iter + 1 < ITER_CNT; iter++)
    {
        // device to host
        HANDLE_ERROR(hipMemcpy(reach[iter], dev_reach[iter], SIZE,
                                hipMemcpyDeviceToHost));

        std::cout << "reachable bits "
                  << iter
                  << " "
                  << countBitsInVolume(reach[iter])
                  << std::endl;
    }

    // teardown
    for (uint32_t iter = 0; iter < ITER_CNT; ++iter)
    {
        HANDLE_ERROR(hipFree(dev_reach[iter]));
        free(reach[iter]);
    }

    HANDLE_ERROR(hipFree(dev_fb));

    return 0;
}
