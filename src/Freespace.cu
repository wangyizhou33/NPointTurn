#include "hip/hip_runtime.h"
#include "Freespace.hpp"
#include <algorithm>
#include <iostream>

Freespace::Freespace()
{
    m_size = m_dim.row * m_dim.col * m_dim.height;
    m_mem  = std::make_unique<uint32_t[]>(m_size);
    std::fill_n(m_mem.get(), m_size, 0u);
}

void Freespace::computeFreespace(const std::vector<Obstacle>& vec)
{
    auto occupy = [this](const Vector2f& v) {
        Vector2ui ind = toIndex(v.x, v.y, m_dim.row, m_dim.col, m_dim.posRes);

        m_mem[index(ind.x, ind.y, 0u, m_dim.row, m_dim.col, m_dim.height)] = 1u;
    };

    // fill the 0-th slice of the volume
    for (const Obstacle& obs : vec)
    {
        for (uint32_t i = 0; i + 1 < obs.boundaryPoints.size(); ++i)
        {
            const Vector2f& v0 = obs.boundaryPoints.at(i);
            const Vector2f& v1 = obs.boundaryPoints.at(i + 1);

            Vector2f e  = v1 - v0;
            float32_t l = e.norm();
            e.normalize();

            for (float32_t d = 0.f; d < l; d += m_dim.posRes)
            {
                Vector2f v = v0 + e * d;
                occupy(v);
            }
            occupy(v1);
        }
    }
    // end of process the 0-th slice

    for (uint32_t k = 1u; k < m_dim.height; ++k)
    {
        computeSlice(k);
    }
}

void Freespace::computeSlice(uint32_t k)
{
    for (uint32_t i = 0u; i < m_dim.row; ++i)
    {
        for (uint32_t j = 0u; j < m_dim.col; ++j)
        {
            Vector2f pos    = toCartesian(i, j, m_dim.row, m_dim.col, m_dim.posRes);
            float32_t theta = static_cast<float32_t>(k) * m_dim.hdgRes;
            uint32_t ind    = index(i, j, k, m_dim.row, m_dim.col, m_dim.height);

            Vector2f newPos = pos.rotate(-theta);

            if (isInBoundary(newPos.x, newPos.y, 30.f)) //TODO: fix the hack
            {
                Vector2ui newInd = toIndex(newPos.x, newPos.y, m_dim.row, m_dim.col, m_dim.posRes);
                if (m_mem[index(newInd.x, newInd.y, 0u, m_dim.row, m_dim.col, m_dim.height)] == 1u)
                {
                    m_mem[ind] = 1u;
                }
            }
        }
    }
}
