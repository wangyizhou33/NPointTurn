#include "hip/hip_runtime.h"
#include "Helper.hpp"

#define N 10

__global__ void add(int* a, int* b, int* c)
{
    int tid = blockIdx.x; // this thread handles the data at its thread id
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(void)
{
    hipDeviceProp_t prop;

    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    for (int i = 0; i < count; i++)
    {
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        printf("   --- General Information for device %d ---\n", i);
        printf("Name:  %s\n", prop.name);
        printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
        printf("Clock rate:  %d\n", prop.clockRate);
        printf("Device copy overlap:  ");
        if (prop.deviceOverlap)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        printf("Kernel execution timeout :  ");
        if (prop.kernelExecTimeoutEnabled)
            printf("Enabled\n");
        else
            printf("Disabled\n");

        printf("   --- Memory Information for device %d ---\n", i);
        printf("Total global mem:  %ld\n", prop.totalGlobalMem);
        printf("Total constant Mem:  %ld\n", prop.totalConstMem);
        printf("Max mem pitch:  %ld\n", prop.memPitch);
        printf("Texture Alignment:  %ld\n", prop.textureAlignment);

        printf("   --- MP Information for device %d ---\n", i);
        printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp:  %d\n", prop.regsPerBlock);
        printf("Threads in warp:  %d\n", prop.warpSize);
        printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0],
               prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0],
               prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }

    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    // display the results
    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // free the memory allocated on the GPU
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));
}
