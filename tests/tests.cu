#include "gtest/gtest.h"
#include "../src/Paper.hpp"
#include <array>

TEST(PaperTests, BitVectorRead)
{
    std::array<uint32_t, 10> cells{};

    EXPECT_EQ(0, bitVectorRead(&cells.front(), 10));

    // put an on bit in the 31st bit
    cells[0] = 1u << 31;
    for (uint32_t i = 0; i < 32; ++i)
    {
        EXPECT_EQ(1 << (31 - i), bitVectorRead(&cells.front(), i));
    }
    for (uint32_t i = 32; i < 64; ++i)
    {
        EXPECT_EQ(0, bitVectorRead(&cells.front(), i));
    }

    // put an on bit in the 32nd bit
    cells[1] = 1 << 0;
    EXPECT_EQ(3, bitVectorRead(&cells.front(), 31));
}

TEST(PaperTests, BitVectorWrite)
{
    std::array<uint32_t, 10> cells{};

    // a random val
    uint32_t val = 1 << 15;

    // what's written should be exactly what's read
    for (uint32_t i = 0; i < 32 * 9; ++i)
    {
        cells.fill(0u); // reset

        bitVectorWrite(&cells.front(), val, i);
        EXPECT_EQ(val, bitVectorRead(&cells.front(), i));
    }

    // write should not update bits outside the range
    cells.fill(0u);
    cells[0] = 1 << 0;
    cells[1] = 1 << 1;

    // write 32 off bits starting from the 1st bit
    bitVectorWrite(&cells.front(), 0, 1);

    // expect the cells unchanged
    EXPECT_EQ(1 << 0, cells[0]);
    EXPECT_EQ(1 << 1, cells[1]);
}

TEST(PaperTests, RaceConditionGPU)
{
    constexpr size_t N = 32 * 32;

    // 4 cells of off bits
    uint32_t *cell, *dev_cell;

    cell = (uint32_t*)malloc(N * sizeof(uint32_t));
    for (uint32_t i = 0; i < N; ++i)
    {
        cell[i] = 0u;
    }

    HANDLE_ERROR(hipMalloc((void**)&dev_cell, N * sizeof(uint32_t)));
    HANDLE_ERROR(hipMemcpy(dev_cell, cell, N * sizeof(uint32_t),
                            hipMemcpyHostToDevice));

    uint32_t offset = 15u; // consistent with writeOnes
    writeOnes<<<1, N>>>(dev_cell, offset);
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy(cell, dev_cell, N * sizeof(uint32_t),
                            hipMemcpyDeviceToHost));

    EXPECT_EQ(4294934528, bitVectorRead(cell, 0)); // 2^32 - 2^offset
    for (uint32_t i = 0; i + 1 < N; ++i)
    {
        uint32_t expect = 4294967295;
        uint32_t actual = bitVectorRead(cell, offset + 32 * i);

        if (expect != actual)
        {
            std::cerr << i << " " << expect << " " << actual << std::endl;
            EXPECT_TRUE(false);
        }
        EXPECT_EQ(4294967295, bitVectorRead(cell, offset + 32 * i));
    }
    EXPECT_EQ(32767, bitVectorRead(cell, 32 * (N - 1))); // 2^offset - 1

    // delete
    HANDLE_ERROR(hipFree(dev_cell));
    free(cell);
}

TEST(PaperTests, TurnCoord1)
{
    // assert every theta slice is a pure translation
    // i.e. turnCoord(x, y, theta + 1) -> turnCoord(x, y, theta) is the same as
    // turnCoord(x+1, y, theta + 1) -> turnCoord(x+1, y, theta)
    // same thing for y

    uint32_t x = X_DIM / 2;
    uint32_t y = Y_DIM / 2;

    for (uint32_t theta = 0; theta < 360; ++theta)
    {

        uint32_t a1 = turnCoord(x, y, theta, X_DIM, Y_DIM, POS_RES, HDG_RES, TURN_R);
        uint32_t a2 = turnCoord(x, y, theta + 1, X_DIM, Y_DIM, POS_RES, HDG_RES, TURN_R);
        uint32_t a  = a2 - a1;

        for (int32_t i = -X_DIM / 2; i <= X_DIM / 2; ++i)
        {
            uint32_t b1 = turnCoord(x + i, y, theta, X_DIM, Y_DIM, POS_RES, HDG_RES, TURN_R);
            uint32_t b2 = turnCoord(x + i, y, theta + 1, X_DIM, Y_DIM, POS_RES, HDG_RES, TURN_R);
            uint32_t b  = b2 - b1;

            EXPECT_EQ(a, b);
        }

        for (int32_t i = -Y_DIM / 2; i <= Y_DIM / 2; ++i)
        {
            uint32_t b1 = turnCoord(x, y + i, theta, X_DIM, Y_DIM, POS_RES, HDG_RES, TURN_R);
            uint32_t b2 = turnCoord(x, y + i, theta + 1, X_DIM, Y_DIM, POS_RES, HDG_RES, TURN_R);
            uint32_t b  = b2 - b1;

            EXPECT_EQ(a, b);
        }
    }
}

TEST(PaperTests, TurnCoord2)
{
    // assert the circular shape of the turn coord
    float32_t x{0.f};
    float32_t y{0.f};
    float32_t tol = 1e-4f;

    for (float32_t theta = 0.f; theta <= 360.f; theta += 1.0f)
    {
        float32_t xout{};
        float32_t yout{};

        // center of the trajectory is (0, TURN_R)
        turnCoord(xout, yout, x, y, theta, TURN_R);
        EXPECT_NEAR(TURN_R * TURN_R, xout * xout + (yout - TURN_R) * (yout - TURN_R), tol);

        // center of the trajectory is (0, -TURN_R)
        turnCoord(xout, yout, x, y, theta, -TURN_R);
        EXPECT_NEAR(TURN_R * TURN_R, xout * xout + (yout + TURN_R) * (yout + TURN_R), tol);
    }
}

TEST(PaperTests, temp)
{
    // std::array<uint32_t, 10> cells{};
    // cells[0] = 4294934528;
    // cells[1] = 4294967295;
    // cells[2] = 4294967295;

    // std::cerr << bitVectorRead(&cells.front(), 0) << std::endl;
    // std::cerr << bitVectorRead(&cells.front(), 1) << std::endl;
    // std::cerr << bitVectorRead(&cells.front(), 2) << std::endl;

    for (uint32_t cr = 0; cr < 32; ++cr)
    {
        std::cout << cr << " " << ~((1u << cr) - 1u) << std::endl;
    }
}

TEST(PaperTests, RaceConditionCPU)
{
    constexpr size_t N = 4;

    // 4 cells of off bits
    uint32_t* cell;

    cell = (uint32_t*)malloc(N * sizeof(uint32_t));
    for (uint32_t i = 0; i < N; ++i)
    {
        cell[i] = 0u;
    }

    uint32_t offset = 15u; // consistent with writeOnes

    bitVectorWrite(cell, 4294967295, offset + 32 * 0);
    bitVectorWrite(cell, 4294967295, offset + 32 * 1);
    bitVectorWrite(cell, 4294967295, offset + 32 * 2);

    EXPECT_EQ(4294967295, bitVectorRead(cell, offset + 32 * 0));
    EXPECT_EQ(4294967295, bitVectorRead(cell, offset + 32 * 1));
    EXPECT_EQ(4294967295, bitVectorRead(cell, offset + 32 * 2));

    // delete
    free(cell);
}

TEST(PaperTests, CountBits)
{
    uint32_t n = (1u << 1) + (1u << 10) + (1u << 15) + (1u << 30);

    EXPECT_EQ(4u, countBits(n));

    n = 4294967295;
    EXPECT_EQ(32u, countBits(n));
}

TEST(PaperTests, Reachability)
{
    uint32_t *dev_reach0, *dev_reach1;
    uint32_t *reach0, *reach1;
    uint32_t* dev_fb;

    HANDLE_ERROR(hipMalloc((void**)&dev_reach0, SIZE));
    HANDLE_ERROR(hipMalloc((void**)&dev_reach1, SIZE));

    HANDLE_ERROR(hipMemset((void*)dev_reach0, 0, SIZE));
    HANDLE_ERROR(hipMemset((void*)dev_reach1, 0, SIZE));

    reach0 = (uint32_t*)malloc(SIZE);
    reach1 = (uint32_t*)malloc(SIZE);

    memset((void*)reach0, 0, SIZE);
    memset((void*)reach1, 0, SIZE);

    HANDLE_ERROR(hipMalloc((void**)&dev_fb, SIZE));
    HANDLE_ERROR(hipMemset((void*)dev_fb, 2147483647, SIZE)); // set all ones

    // set reach0
    uint32_t middle = turnCoord(X_DIM / 2, Y_DIM / 2, 0,
                                X_DIM, Y_DIM, POS_RES, HDG_RES, TURN_R);

    bitVectorWrite(reach0, 4294967295, middle);

    HANDLE_ERROR(hipMemcpy(dev_reach0, reach0, SIZE,
                            hipMemcpyHostToDevice));

    TIME_PRINT("sweep ",
               bitSweepTurn(dev_reach1,
                            dev_fb,
                            dev_reach0,
                            TURN_R,
                            nullptr);
               hipDeviceSynchronize(););

    HANDLE_ERROR(hipGetLastError());

    HANDLE_ERROR(hipMemcpy(reach1, dev_reach1, SIZE,
                            hipMemcpyDeviceToHost));

    // assert each theta slice has 32 ON-bits
    for (uint32_t theta = 0; theta < 360; ++theta)
    {
        uint32_t startIndex = X_DIM * Y_DIM * theta;
        uint32_t endIndex   = X_DIM * Y_DIM * (theta + 1);

        uint32_t reachableBitCount = 0u;

        for (uint32_t coordIndex = startIndex / 32; coordIndex < endIndex / 32; coordIndex++)
        {
            reachableBitCount += countBits(reach1[coordIndex]);
        }
        if (reachableBitCount != 32)
            std::cerr << " theta " << theta
                      << " reachable bits " << reachableBitCount << std::endl;
    }

    // delete
    HANDLE_ERROR(hipFree(dev_reach0));
    HANDLE_ERROR(hipFree(dev_reach1));
    HANDLE_ERROR(hipFree(dev_fb));

    free(reach0);
    free(reach1);
}

TEST(PaperTests, Obstacle)
{
    uint32_t *dev_reach0, *dev_reach1;
    uint32_t *reach0, *reach1;
    uint32_t* dev_fb;

    HANDLE_ERROR(hipMalloc((void**)&dev_reach0, SIZE));
    HANDLE_ERROR(hipMalloc((void**)&dev_reach1, SIZE));

    HANDLE_ERROR(hipMemset((void*)dev_reach0, 0, SIZE));
    HANDLE_ERROR(hipMemset((void*)dev_reach1, 0, SIZE));

    reach0 = (uint32_t*)malloc(SIZE);
    reach1 = (uint32_t*)malloc(SIZE);

    memset((void*)reach0, 0, SIZE);
    memset((void*)reach1, 0, SIZE);

    HANDLE_ERROR(hipMalloc((void**)&dev_fb, SIZE));
    HANDLE_ERROR(hipMemset((void*)dev_fb, 2147483647, SIZE / 2)); // set ones for half of the theta slices

    // set reach0
    uint32_t middle = turnCoord(X_DIM / 2, Y_DIM / 2, 0,
                                X_DIM, Y_DIM, POS_RES, HDG_RES, TURN_R);

    bitVectorWrite(reach0, 4294967295, middle);

    HANDLE_ERROR(hipMemcpy(dev_reach0, reach0, SIZE,
                            hipMemcpyHostToDevice));

    TIME_PRINT("sweep ",
               bitSweepTurn(dev_reach1,
                            dev_fb,
                            dev_reach0,
                            TURN_R,
                            nullptr);
               hipDeviceSynchronize(););

    HANDLE_ERROR(hipGetLastError());

    HANDLE_ERROR(hipMemcpy(reach1, dev_reach1, SIZE,
                            hipMemcpyDeviceToHost));

    // assert bits are ON at theta = 179
    uint32_t theta      = 179;
    uint32_t startIndex = X_DIM * Y_DIM * theta;
    uint32_t endIndex   = X_DIM * Y_DIM * (theta + 1);

    uint32_t reachableBitCount = 0u;
    for (uint32_t coordIndex = startIndex / 32; coordIndex < endIndex / 32; coordIndex++)
    {
        reachableBitCount += countBits(reach1[coordIndex]);
    }
    EXPECT_EQ(32, reachableBitCount);

    // assert bits are OFF at theta = 180
    theta      = 180;
    startIndex = endIndex;
    endIndex   = X_DIM * Y_DIM * (theta + 1);

    reachableBitCount = 0u;
    for (uint32_t coordIndex = startIndex / 32; coordIndex < endIndex / 32; coordIndex++)
    {
        reachableBitCount += countBits(reach1[coordIndex]);
    }
    EXPECT_EQ(0, reachableBitCount);

    // delete
    HANDLE_ERROR(hipFree(dev_reach0));
    HANDLE_ERROR(hipFree(dev_reach1));
    HANDLE_ERROR(hipFree(dev_fb));

    free(reach0);
    free(reach1);
}

TEST(PaperTests, shuffle)
{
    constexpr size_t N = 8;

    // 4 cells of off bits
    uint32_t *cell, *dev_cell;

    cell = (uint32_t*)malloc(N * sizeof(uint32_t));
    for (uint32_t i = 0; i < N; ++i)
    {
        cell[i] = 0u;
    }

    HANDLE_ERROR(hipMalloc((void**)&dev_cell, N * sizeof(uint32_t)));
    HANDLE_ERROR(hipMemcpy(dev_cell, cell, N * sizeof(uint32_t),
                            hipMemcpyHostToDevice));

    shuffle<<<1, N>>>(dev_cell);
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy(cell, dev_cell, N * sizeof(uint32_t),
                            hipMemcpyDeviceToHost));

    for (uint32_t i = 1u; i < N; ++i)
    {
        EXPECT_EQ(1 << (i - 1u), cell[i]);
    }
    EXPECT_EQ((1 << 7), cell[0]);
}

int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}